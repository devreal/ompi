#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023 The University of Tennessee and The University
 *                         of Tennessee Research Foundation.  All rights
 *                         reserved.
 * Copyright (c) 2020      Research Organization for Information Science
 *                         and Technology (RIST).  All rights reserved.
 * $COPYRIGHT$
 *
 * Additional copyrights may follow
 *
 * $HEADER$
 */

#include <sys/types.h>

#include <hip/hip_complex.h>

#include "op_cuda_impl.h"

/* TODO: missing support for
 * - short float (conditional on whether short float is available)
 * - complex
 * - 3buff implementation
 */

#define THREADS_PER_BLOCK 512

#define OP_FUNC(name, type_name, type, op)                                                          \
    static __global__ void                                                                          \
    ompi_op_cuda_2buff_##name##_##type_name##_kernel(const type *in, type *inout, int n) {          \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                    \
        const int stride = blockDim.x * gridDim.x;                                                  \
        for (int i = index; i < n; i += stride) {                                                   \
            inout[i] = inout[i] op in[i];                                                           \
        }                                                                                           \
    }                                                                                               \
    void ompi_op_cuda_2buff_##name##_##type_name##_submit(const type *in,                           \
                                                   type *inout,                                     \
                                                   int count,                                       \
                                                   int threads_per_block,                           \
                                                   hipStream_t stream) {                               \
        int threads = threads_per_block;                                                            \
        int blocks  = (count + threads-1) / threads;                                                \
        int n = count;                                                                              \
        hipStream_t s = stream;                                                                        \
        ompi_op_cuda_2buff_##name##_##type_name##_kernel<<<blocks, threads, 0, s>>>(in, inout, n);  \
    }


#define FUNC_FUNC(name, type_name, type)                                                            \
    static __global__ void                                                                          \
    ompi_op_cuda_2buff_##name##_##type_name##_kernel(const type *in, type *inout, int n) {          \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                    \
        const int stride = blockDim.x * gridDim.x;                                                  \
        for (int i = index; i < n; i += stride) {                                                   \
            inout[i] = current_func(inout[i], in[i]);                                               \
        }                                                                                           \
    }                                                                                               \
    void                                                                                            \
    ompi_op_cuda_2buff_##name##_##type_name##_submit(const type *in,                                \
                                              type *inout,                                          \
                                              int count,                                            \
                                              int threads_per_block,                                \
                                              hipStream_t stream) {                                    \
        int threads = threads_per_block;                                                            \
        int blocks  = (count + threads-1) / threads;                                                \
        int n = count;                                                                              \
        hipStream_t s = stream;                                                                        \
        ompi_op_cuda_2buff_##name##_##type_name##_kernel<<<blocks, threads, 0, s>>>(in, inout, n);  \
    }

/*
 * Since all the functions in this file are essentially identical, we
 * use a macro to substitute in names and types.  The core operation
 * in all functions that use this macro is the same.
 *
 * This macro is for minloc and maxloc
 */

#define LOC_FUNC(name, type_name, op)                                                               \
    static __global__ void                                                                          \
    ompi_op_cuda_2buff_##name##_##type_name##_kernel(const ompi_op_predefined_##type_name##_t *in,  \
                                                     ompi_op_predefined_##type_name##_t *inout,     \
                                                     int n)                                         \
    {                                                                                               \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                    \
        const int stride = blockDim.x * gridDim.x;                                                  \
        for (int i = index; i < n; i += stride) {                                                   \
            const ompi_op_predefined_##type_name##_t *a = &in[i];                                   \
            ompi_op_predefined_##type_name##_t *b = &inout[i];                                      \
            if (a->v op b->v) {                                                                     \
                b->v = a->v;                                                                        \
                b->k = a->k;                                                                        \
            } else if (a->v == b->v) {                                                              \
                b->k = (b->k < a->k ? b->k : a->k);                                                 \
            }                                                                                       \
        }                                                                                           \
    }                                                                                               \
    void                                                                                            \
    ompi_op_cuda_2buff_##name##_##type_name##_submit(const ompi_op_predefined_##type_name##_t *a,   \
                                            ompi_op_predefined_##type_name##_t *b,                  \
                                            int count,                                              \
                                            int threads_per_block,                                  \
                                            hipStream_t stream) {                                      \
        int threads = threads_per_block;                                                            \
        int blocks  = (count + threads-1) / threads;                                                \
        hipStream_t s = stream;                                                                        \
        ompi_op_cuda_2buff_##name##_##type_name##_kernel<<<blocks, threads, 0, s>>>(a, b, count);   \
    }

/*************************************************************************
 * Max
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) > (b) ? (a) : (b))
/* C integer */
FUNC_FUNC(max,   int8_t,   int8_t)
FUNC_FUNC(max,  uint8_t,  uint8_t)
FUNC_FUNC(max,  int16_t,  int16_t)
FUNC_FUNC(max, uint16_t, uint16_t)
FUNC_FUNC(max,  int32_t,  int32_t)
FUNC_FUNC(max, uint32_t, uint32_t)
FUNC_FUNC(max,  int64_t,  int64_t)
FUNC_FUNC(max, uint64_t, uint64_t)
FUNC_FUNC(max,  long,  long)
FUNC_FUNC(max,  unsigned_long, unsigned long)

FUNC_FUNC(max, float, float)
FUNC_FUNC(max, double, double)
FUNC_FUNC(max, long_double, long double)

/*************************************************************************
 * Min
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) < (b) ? (a) : (b))
/* C integer */
FUNC_FUNC(min,   int8_t,   int8_t)
FUNC_FUNC(min,  uint8_t,  uint8_t)
FUNC_FUNC(min,  int16_t,  int16_t)
FUNC_FUNC(min, uint16_t, uint16_t)
FUNC_FUNC(min,  int32_t,  int32_t)
FUNC_FUNC(min, uint32_t, uint32_t)
FUNC_FUNC(min,  int64_t,  int64_t)
FUNC_FUNC(min, uint64_t, uint64_t)
FUNC_FUNC(min,  long,  long)
FUNC_FUNC(min,  unsigned_long, unsigned long)


FUNC_FUNC(min, float, float)
FUNC_FUNC(min, double, double)
FUNC_FUNC(min, long_double, long double)

/*************************************************************************
 * Sum
 *************************************************************************/

/* C integer */
OP_FUNC(sum,   int8_t,   int8_t, +=)
OP_FUNC(sum,  uint8_t,  uint8_t, +=)
OP_FUNC(sum,  int16_t,  int16_t, +=)
OP_FUNC(sum, uint16_t, uint16_t, +=)
OP_FUNC(sum,  int32_t,  int32_t, +=)
OP_FUNC(sum, uint32_t, uint32_t, +=)
OP_FUNC(sum,  int64_t,  int64_t, +=)
OP_FUNC(sum, uint64_t, uint64_t, +=)
OP_FUNC(sum,  long,  long, +=)
OP_FUNC(sum,  unsigned_long, unsigned long, +=)

OP_FUNC(sum, float, float, +=)
OP_FUNC(sum, double, double, +=)
OP_FUNC(sum, long_double, long double, +=)

/* Complex */
#if 0
#if defined(HAVE_SHORT_FLOAT__COMPLEX)
OP_FUNC(sum, c_short_float_complex, short float _Complex, +=)
#elif defined(HAVE_OPAL_SHORT_FLOAT_COMPLEX_T)
COMPLEX_SUM_FUNC(c_short_float_complex, opal_short_float_t)
#endif
#endif // 0
#undef current_func
#define current_func(a, b) (hipCmulf(a,b))
FUNC_FUNC(sum, c_float_complex, hipFloatComplex)
#undef current_func
#define current_func(a, b) (hipCmul(a,b))
FUNC_FUNC(sum, c_double_complex, hipDoubleComplex)
//OP_FUNC(sum, c_long_double_complex, cuLongDoubleComplex, +=)

/*************************************************************************
 * Product
 *************************************************************************/

/* C integer */
OP_FUNC(prod,   int8_t,   int8_t, *=)
OP_FUNC(prod,  uint8_t,  uint8_t, *=)
OP_FUNC(prod,  int16_t,  int16_t, *=)
OP_FUNC(prod, uint16_t, uint16_t, *=)
OP_FUNC(prod,  int32_t,  int32_t, *=)
OP_FUNC(prod, uint32_t, uint32_t, *=)
OP_FUNC(prod,  int64_t,  int64_t, *=)
OP_FUNC(prod, uint64_t, uint64_t, *=)
OP_FUNC(prod,  long,  long, *=)
OP_FUNC(prod,  unsigned_long, unsigned long, *=)

OP_FUNC(prod, float, float, *=)
OP_FUNC(prod, double, double, *=)
OP_FUNC(prod, long_double, long double, *=)

/* Complex */
#if 0
#if defined(HAVE_SHORT_FLOAT__COMPLEX)
OP_FUNC(prod, c_short_float_complex, short float _Complex, *=)
#elif defined(HAVE_OPAL_SHORT_FLOAT_COMPLEX_T)
COMPLEX_PROD_FUNC(c_short_float_complex, opal_short_float_t)
#endif
OP_FUNC(prod, c_float_complex, float _Complex, *=)
OP_FUNC(prod, c_double_complex, double _Complex, *=)
OP_FUNC(prod, c_long_double_complex, long double _Complex, *=)
#endif // 0

/*************************************************************************
 * Logical AND
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) && (b))
/* C integer */
FUNC_FUNC(land,   int8_t,   int8_t)
FUNC_FUNC(land,  uint8_t,  uint8_t)
FUNC_FUNC(land,  int16_t,  int16_t)
FUNC_FUNC(land, uint16_t, uint16_t)
FUNC_FUNC(land,  int32_t,  int32_t)
FUNC_FUNC(land, uint32_t, uint32_t)
FUNC_FUNC(land,  int64_t,  int64_t)
FUNC_FUNC(land, uint64_t, uint64_t)
FUNC_FUNC(land,  long,  long)
FUNC_FUNC(land,  unsigned_long, unsigned long)

/* Logical */
#if OMPI_HAVE_FORTRAN_LOGICAL
FUNC_FUNC(land, fortran_logical, ompi_fortran_logical_t)
#endif
/* C++ bool */
FUNC_FUNC(land, bool, bool)

/*************************************************************************
 * Logical OR
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) || (b))
/* C integer */
FUNC_FUNC(lor,   int8_t,   int8_t)
FUNC_FUNC(lor,  uint8_t,  uint8_t)
FUNC_FUNC(lor,  int16_t,  int16_t)
FUNC_FUNC(lor, uint16_t, uint16_t)
FUNC_FUNC(lor,  int32_t,  int32_t)
FUNC_FUNC(lor, uint32_t, uint32_t)
FUNC_FUNC(lor,  int64_t,  int64_t)
FUNC_FUNC(lor, uint64_t, uint64_t)
FUNC_FUNC(lor,  long,  long)
FUNC_FUNC(lor,  unsigned_long, unsigned long)

/* C++ bool */
FUNC_FUNC(lor, bool, bool)

/*************************************************************************
 * Logical XOR
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a ? 1 : 0) ^ (b ? 1: 0))
/* C integer */
FUNC_FUNC(lxor,   int8_t,   int8_t)
FUNC_FUNC(lxor,  uint8_t,  uint8_t)
FUNC_FUNC(lxor,  int16_t,  int16_t)
FUNC_FUNC(lxor, uint16_t, uint16_t)
FUNC_FUNC(lxor,  int32_t,  int32_t)
FUNC_FUNC(lxor, uint32_t, uint32_t)
FUNC_FUNC(lxor,  int64_t,  int64_t)
FUNC_FUNC(lxor, uint64_t, uint64_t)
FUNC_FUNC(lxor,  long,  long)
FUNC_FUNC(lxor,  unsigned_long, unsigned long)

/* C++ bool */
FUNC_FUNC(lxor, bool, bool)

/*************************************************************************
 * Bitwise AND
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) & (b))
/* C integer */
FUNC_FUNC(band,   int8_t,   int8_t)
FUNC_FUNC(band,  uint8_t,  uint8_t)
FUNC_FUNC(band,  int16_t,  int16_t)
FUNC_FUNC(band, uint16_t, uint16_t)
FUNC_FUNC(band,  int32_t,  int32_t)
FUNC_FUNC(band, uint32_t, uint32_t)
FUNC_FUNC(band,  int64_t,  int64_t)
FUNC_FUNC(band, uint64_t, uint64_t)
FUNC_FUNC(band,  long,  long)
FUNC_FUNC(band,  unsigned_long, unsigned long)

/* Byte */
FUNC_FUNC(band, byte, char)

/*************************************************************************
 * Bitwise OR
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) | (b))
/* C integer */
FUNC_FUNC(bor,   int8_t,   int8_t)
FUNC_FUNC(bor,  uint8_t,  uint8_t)
FUNC_FUNC(bor,  int16_t,  int16_t)
FUNC_FUNC(bor, uint16_t, uint16_t)
FUNC_FUNC(bor,  int32_t,  int32_t)
FUNC_FUNC(bor, uint32_t, uint32_t)
FUNC_FUNC(bor,  int64_t,  int64_t)
FUNC_FUNC(bor, uint64_t, uint64_t)
FUNC_FUNC(bor,  long,  long)
FUNC_FUNC(bor,  unsigned_long, unsigned long)

/* Byte */
FUNC_FUNC(bor, byte, char)

/*************************************************************************
 * Bitwise XOR
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) ^ (b))
/* C integer */
FUNC_FUNC(bxor,   int8_t,   int8_t)
FUNC_FUNC(bxor,  uint8_t,  uint8_t)
FUNC_FUNC(bxor,  int16_t,  int16_t)
FUNC_FUNC(bxor, uint16_t, uint16_t)
FUNC_FUNC(bxor,  int32_t,  int32_t)
FUNC_FUNC(bxor, uint32_t, uint32_t)
FUNC_FUNC(bxor,  int64_t,  int64_t)
FUNC_FUNC(bxor, uint64_t, uint64_t)
FUNC_FUNC(bxor,  long,  long)
FUNC_FUNC(bxor,  unsigned_long, unsigned long)

/* Byte */
FUNC_FUNC(bxor, byte, char)

/*************************************************************************
 * Max location
 *************************************************************************/

LOC_FUNC(maxloc, float_int, >)
LOC_FUNC(maxloc, double_int, >)
LOC_FUNC(maxloc, long_int, >)
LOC_FUNC(maxloc, 2int, >)
LOC_FUNC(maxloc, short_int, >)
LOC_FUNC(maxloc, long_double_int, >)

/*************************************************************************
 * Min location
 *************************************************************************/

LOC_FUNC(minloc, float_int, <)
LOC_FUNC(minloc, double_int, <)
LOC_FUNC(minloc, long_int, <)
LOC_FUNC(minloc, 2int, <)
LOC_FUNC(minloc, short_int, <)
LOC_FUNC(minloc, long_double_int, <)


/*
 *  This is a three buffer (2 input and 1 output) version of the reduction
 *    routines, needed for some optimizations.
 */
#define OP_FUNC_3BUF(name, type_name, type, op)                                                     \
    static __global__ void                                                                          \
    ompi_op_cuda_3buff_##name##_##type_name##_kernel(const type *in1, const type* in2,              \
                                                     type *out, int n) {                            \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                    \
        const int stride = blockDim.x * gridDim.x;                                                  \
        for (int i = index; i < n; i += stride) {                                                   \
            out[i] = in1[i] op in2[i];                                                              \
        }                                                                                           \
    }                                                                                               \
    void ompi_op_cuda_3buff_##name##_##type_name##_submit(const type *in1, const type *in2,         \
                                                          type *out, int count,                     \
                                                          int threads_per_block,                    \
                                                          hipStream_t stream) {                        \
        int threads = threads_per_block;                                                            \
        int blocks  = (count+threads-1) / threads;                                                  \
        ompi_op_cuda_3buff_##name##_##type_name##_kernel<<<blocks, threads,                         \
                                                           0, stream>>>(in1, in2, out, count);      \
    }


/*
 * Since all the functions in this file are essentially identical, we
 * use a macro to substitute in names and types.  The core operation
 * in all functions that use this macro is the same.
 *
 * This macro is for (out = op(in1, in2))
 */
#define FUNC_FUNC_3BUF(name, type_name, type)                                                       \
    static __global__ void                                                                          \
    ompi_op_cuda_3buff_##name##_##type_name##_kernel(const type *in1, const type *in2,              \
                                                     type *out, int n) {                            \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                    \
        const int stride = blockDim.x * gridDim.x;                                                  \
        for (int i = index; i < n; i += stride) {                                                   \
            out[i] = current_func(in1[i], in2[i]);                                                  \
        }                                                                                           \
    }                                                                                               \
    void                                                                                            \
    ompi_op_cuda_3buff_##name##_##type_name##_submit(const type *in1, const type *in2,              \
                                                     type *out, int count,                          \
                                                     int threads_per_block,                         \
                                                     hipStream_t stream) {                             \
        int threads = threads_per_block;                                                            \
        int blocks  = (count+threads-1) / threads;                                                  \
        ompi_op_cuda_3buff_##name##_##type_name##_kernel<<<blocks, threads,                         \
                                                           0, stream>>>(in1, in2, out, count);      \
    }

/*
 * Since all the functions in this file are essentially identical, we
 * use a macro to substitute in names and types.  The core operation
 * in all functions that use this macro is the same.
 *
 * This macro is for minloc and maxloc
 */
/*
#define LOC_STRUCT(type_name, type1, type2) \
  typedef struct { \
      type1 v; \
      type2 k; \
  } ompi_op_predefined_##type_name##_t;
*/

#define LOC_FUNC_3BUF(name, type_name, op)                                                          \
    static __global__ void                                                                          \
    ompi_op_cuda_3buff_##name##_##type_name##_kernel(const ompi_op_predefined_##type_name##_t *in1, \
                                                     const ompi_op_predefined_##type_name##_t *in2, \
                                                     ompi_op_predefined_##type_name##_t *out,       \
                                                     int n)                                         \
    {                                                                                               \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                    \
        const int stride = blockDim.x * gridDim.x;                                                  \
        for (int i = index; i < n; i += stride) {                                                   \
            const ompi_op_predefined_##type_name##_t *a1 = &in1[i];                                 \
            const ompi_op_predefined_##type_name##_t *a2 = &in2[i];                                 \
            ompi_op_predefined_##type_name##_t *b = &out[i];                                        \
            if (a1->v op a2->v) {                                                                   \
                b->v = a1->v;                                                                       \
                b->k = a1->k;                                                                       \
            } else if (a1->v == a2->v) {                                                            \
                b->v = a1->v;                                                                       \
                b->k = (a2->k < a1->k ? a2->k : a1->k);                                             \
            } else {                                                                                \
                b->v = a2->v;                                                                       \
                b->k = a2->k;                                                                       \
            }                                                                                       \
        }                                                                                           \
    }                                                                                               \
    void                                                                                            \
    ompi_op_cuda_3buff_##name##_##type_name##_submit(const ompi_op_predefined_##type_name##_t *in1, \
                                                     const ompi_op_predefined_##type_name##_t *in2, \
                                                     ompi_op_predefined_##type_name##_t *out,       \
                                                     int count,                                     \
                                                     int threads_per_block,                         \
                                                     hipStream_t stream)                               \
    {                                                                                               \
        int threads = threads_per_block;                                                            \
        int blocks  = (count+threads-1) / threads;                                                  \
        ompi_op_cuda_3buff_##name##_##type_name##_kernel<<<blocks, threads,                         \
                                                           0, stream>>>(in1, in2, out, count);      \
    }


/*************************************************************************
 * Max
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) > (b) ? (a) : (b))
/* C integer */
FUNC_FUNC_3BUF(max,   int8_t,   int8_t)
FUNC_FUNC_3BUF(max,  uint8_t,  uint8_t)
FUNC_FUNC_3BUF(max,  int16_t,  int16_t)
FUNC_FUNC_3BUF(max, uint16_t, uint16_t)
FUNC_FUNC_3BUF(max,  int32_t,  int32_t)
FUNC_FUNC_3BUF(max, uint32_t, uint32_t)
FUNC_FUNC_3BUF(max,  int64_t,  int64_t)
FUNC_FUNC_3BUF(max, uint64_t, uint64_t)
FUNC_FUNC_3BUF(max,  long,  long)
FUNC_FUNC_3BUF(max,  unsigned_long, unsigned long)

/* Fortran integer */
#if OMPI_HAVE_FORTRAN_INTEGER
FUNC_FUNC_3BUF(max, fortran_integer, ompi_fortran_integer_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER1
FUNC_FUNC_3BUF(max, fortran_integer1, ompi_fortran_integer1_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER2
FUNC_FUNC_3BUF(max, fortran_integer2, ompi_fortran_integer2_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER4
FUNC_FUNC_3BUF(max, fortran_integer4, ompi_fortran_integer4_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER8
FUNC_FUNC_3BUF(max, fortran_integer8, ompi_fortran_integer8_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER16
FUNC_FUNC_3BUF(max, fortran_integer16, ompi_fortran_integer16_t)
#endif
/* Floating point */
#if defined(HAVE_SHORT_FLOAT)
FUNC_FUNC_3BUF(max, short_float, short float)
#elif defined(HAVE_OPAL_SHORT_FLOAT_T)
FUNC_FUNC_3BUF(max, short_float, opal_short_float_t)
#endif
FUNC_FUNC_3BUF(max, float, float)
FUNC_FUNC_3BUF(max, double, double)
FUNC_FUNC_3BUF(max, long_double, long double)
#if OMPI_HAVE_FORTRAN_REAL
FUNC_FUNC_3BUF(max, fortran_real, ompi_fortran_real_t)
#endif
#if OMPI_HAVE_FORTRAN_DOUBLE_PRECISION
FUNC_FUNC_3BUF(max, fortran_double_precision, ompi_fortran_double_precision_t)
#endif
#if OMPI_HAVE_FORTRAN_REAL2
FUNC_FUNC_3BUF(max, fortran_real2, ompi_fortran_real2_t)
#endif
#if OMPI_HAVE_FORTRAN_REAL4
FUNC_FUNC_3BUF(max, fortran_real4, ompi_fortran_real4_t)
#endif
#if OMPI_HAVE_FORTRAN_REAL8
FUNC_FUNC_3BUF(max, fortran_real8, ompi_fortran_real8_t)
#endif
#if OMPI_HAVE_FORTRAN_REAL16 && OMPI_REAL16_MATCHES_C
FUNC_FUNC_3BUF(max, fortran_real16, ompi_fortran_real16_t)
#endif


/*************************************************************************
 * Min
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) < (b) ? (a) : (b))
/* C integer */
FUNC_FUNC_3BUF(min,   int8_t,   int8_t)
FUNC_FUNC_3BUF(min,  uint8_t,  uint8_t)
FUNC_FUNC_3BUF(min,  int16_t,  int16_t)
FUNC_FUNC_3BUF(min, uint16_t, uint16_t)
FUNC_FUNC_3BUF(min,  int32_t,  int32_t)
FUNC_FUNC_3BUF(min, uint32_t, uint32_t)
FUNC_FUNC_3BUF(min,  int64_t,  int64_t)
FUNC_FUNC_3BUF(min, uint64_t, uint64_t)
FUNC_FUNC_3BUF(min,  long,  long)
FUNC_FUNC_3BUF(min,  unsigned_long, unsigned long)

/* Fortran integer */
#if OMPI_HAVE_FORTRAN_INTEGER
FUNC_FUNC_3BUF(min, fortran_integer, ompi_fortran_integer_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER1
FUNC_FUNC_3BUF(min, fortran_integer1, ompi_fortran_integer1_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER2
FUNC_FUNC_3BUF(min, fortran_integer2, ompi_fortran_integer2_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER4
FUNC_FUNC_3BUF(min, fortran_integer4, ompi_fortran_integer4_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER8
FUNC_FUNC_3BUF(min, fortran_integer8, ompi_fortran_integer8_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER16
FUNC_FUNC_3BUF(min, fortran_integer16, ompi_fortran_integer16_t)
#endif
/* Floating point */
#if defined(HAVE_SHORT_FLOAT)
FUNC_FUNC_3BUF(min, short_float, short float)
#elif defined(HAVE_OPAL_SHORT_FLOAT_T)
FUNC_FUNC_3BUF(min, short_float, opal_short_float_t)
#endif
FUNC_FUNC_3BUF(min, float, float)
FUNC_FUNC_3BUF(min, double, double)
FUNC_FUNC_3BUF(min, long_double, long double)
#if OMPI_HAVE_FORTRAN_REAL
FUNC_FUNC_3BUF(min, fortran_real, ompi_fortran_real_t)
#endif
#if OMPI_HAVE_FORTRAN_DOUBLE_PRECISION
FUNC_FUNC_3BUF(min, fortran_double_precision, ompi_fortran_double_precision_t)
#endif
#if OMPI_HAVE_FORTRAN_REAL2
FUNC_FUNC_3BUF(min, fortran_real2, ompi_fortran_real2_t)
#endif
#if OMPI_HAVE_FORTRAN_REAL4
FUNC_FUNC_3BUF(min, fortran_real4, ompi_fortran_real4_t)
#endif
#if OMPI_HAVE_FORTRAN_REAL8
FUNC_FUNC_3BUF(min, fortran_real8, ompi_fortran_real8_t)
#endif
#if OMPI_HAVE_FORTRAN_REAL16 && OMPI_REAL16_MATCHES_C
FUNC_FUNC_3BUF(min, fortran_real16, ompi_fortran_real16_t)
#endif

/*************************************************************************
 * Sum
 *************************************************************************/

/* C integer */
OP_FUNC_3BUF(sum,   int8_t,   int8_t, +)
OP_FUNC_3BUF(sum,  uint8_t,  uint8_t, +)
OP_FUNC_3BUF(sum,  int16_t,  int16_t, +)
OP_FUNC_3BUF(sum, uint16_t, uint16_t, +)
OP_FUNC_3BUF(sum,  int32_t,  int32_t, +)
OP_FUNC_3BUF(sum, uint32_t, uint32_t, +)
OP_FUNC_3BUF(sum,  int64_t,  int64_t, +)
OP_FUNC_3BUF(sum, uint64_t, uint64_t, +)
OP_FUNC_3BUF(sum,  long,  long, +)
OP_FUNC_3BUF(sum,  unsigned_long, unsigned long, +)

/* Fortran integer */
#if OMPI_HAVE_FORTRAN_INTEGER
OP_FUNC_3BUF(sum, fortran_integer, ompi_fortran_integer_t, +)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER1
OP_FUNC_3BUF(sum, fortran_integer1, ompi_fortran_integer1_t, +)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER2
OP_FUNC_3BUF(sum, fortran_integer2, ompi_fortran_integer2_t, +)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER4
OP_FUNC_3BUF(sum, fortran_integer4, ompi_fortran_integer4_t, +)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER8
OP_FUNC_3BUF(sum, fortran_integer8, ompi_fortran_integer8_t, +)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER16
OP_FUNC_3BUF(sum, fortran_integer16, ompi_fortran_integer16_t, +)
#endif
/* Floating point */
#if defined(HAVE_SHORT_FLOAT)
OP_FUNC_3BUF(sum, short_float, short float, +)
#elif defined(HAVE_OPAL_SHORT_FLOAT_T)
OP_FUNC_3BUF(sum, short_float, opal_short_float_t, +)
#endif
OP_FUNC_3BUF(sum, float, float, +)
OP_FUNC_3BUF(sum, double, double, +)
OP_FUNC_3BUF(sum, long_double, long double, +)
#if OMPI_HAVE_FORTRAN_REAL
OP_FUNC_3BUF(sum, fortran_real, ompi_fortran_real_t, +)
#endif
#if OMPI_HAVE_FORTRAN_DOUBLE_PRECISION
OP_FUNC_3BUF(sum, fortran_double_precision, ompi_fortran_double_precision_t, +)
#endif
#if OMPI_HAVE_FORTRAN_REAL2
OP_FUNC_3BUF(sum, fortran_real2, ompi_fortran_real2_t, +)
#endif
#if OMPI_HAVE_FORTRAN_REAL4
OP_FUNC_3BUF(sum, fortran_real4, ompi_fortran_real4_t, +)
#endif
#if OMPI_HAVE_FORTRAN_REAL8
OP_FUNC_3BUF(sum, fortran_real8, ompi_fortran_real8_t, +)
#endif
#if OMPI_HAVE_FORTRAN_REAL16 && OMPI_REAL16_MATCHES_C
OP_FUNC_3BUF(sum, fortran_real16, ompi_fortran_real16_t, +)
#endif
/* Complex */
#if 0
#if defined(HAVE_SHORT_FLOAT__COMPLEX)
OP_FUNC_3BUF(sum, c_short_float_complex, short float _Complex, +)
#elif defined(HAVE_OPAL_SHORT_FLOAT_COMPLEX_T)
COMPLEX_SUM_FUNC_3BUF(c_short_float_complex, opal_short_float_t)
#endif
#endif // 0
#undef current_func
#define current_func(a, b) (hipCmulf(a,b))
FUNC_FUNC_3BUF(sum, c_float_complex, hipFloatComplex)
#undef current_func
#define current_func(a, b) (hipCmul(a,b))
FUNC_FUNC_3BUF(sum, c_double_complex, hipDoubleComplex)
//OP_FUNC_3BUF(sum, c_long_double_complex, cuLongDoubleComplex, +)

/*************************************************************************
 * Product
 *************************************************************************/

/* C integer */
OP_FUNC_3BUF(prod,   int8_t,   int8_t, *)
OP_FUNC_3BUF(prod,  uint8_t,  uint8_t, *)
OP_FUNC_3BUF(prod,  int16_t,  int16_t, *)
OP_FUNC_3BUF(prod, uint16_t, uint16_t, *)
OP_FUNC_3BUF(prod,  int32_t,  int32_t, *)
OP_FUNC_3BUF(prod, uint32_t, uint32_t, *)
OP_FUNC_3BUF(prod,  int64_t,  int64_t, *)
OP_FUNC_3BUF(prod, uint64_t, uint64_t, *)
OP_FUNC_3BUF(prod,  long,  long, *)
OP_FUNC_3BUF(prod,  unsigned_long, unsigned long, *)

/* Fortran integer */
#if OMPI_HAVE_FORTRAN_INTEGER
OP_FUNC_3BUF(prod, fortran_integer, ompi_fortran_integer_t, *)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER1
OP_FUNC_3BUF(prod, fortran_integer1, ompi_fortran_integer1_t, *)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER2
OP_FUNC_3BUF(prod, fortran_integer2, ompi_fortran_integer2_t, *)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER4
OP_FUNC_3BUF(prod, fortran_integer4, ompi_fortran_integer4_t, *)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER8
OP_FUNC_3BUF(prod, fortran_integer8, ompi_fortran_integer8_t, *)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER16
OP_FUNC_3BUF(prod, fortran_integer16, ompi_fortran_integer16_t, *)
#endif
/* Floating point */
#if defined(HAVE_SHORT_FLOAT)
OP_FUNC_3BUF(prod, short_float, short float, *)
#elif defined(HAVE_OPAL_SHORT_FLOAT_T)
OP_FUNC_3BUF(prod, short_float, opal_short_float_t, *)
#endif
OP_FUNC_3BUF(prod, float, float, *)
OP_FUNC_3BUF(prod, double, double, *)
OP_FUNC_3BUF(prod, long_double, long double, *)
#if OMPI_HAVE_FORTRAN_REAL
OP_FUNC_3BUF(prod, fortran_real, ompi_fortran_real_t, *)
#endif
#if OMPI_HAVE_FORTRAN_DOUBLE_PRECISION
OP_FUNC_3BUF(prod, fortran_double_precision, ompi_fortran_double_precision_t, *)
#endif
#if OMPI_HAVE_FORTRAN_REAL2
OP_FUNC_3BUF(prod, fortran_real2, ompi_fortran_real2_t, *)
#endif
#if OMPI_HAVE_FORTRAN_REAL4
OP_FUNC_3BUF(prod, fortran_real4, ompi_fortran_real4_t, *)
#endif
#if OMPI_HAVE_FORTRAN_REAL8
OP_FUNC_3BUF(prod, fortran_real8, ompi_fortran_real8_t, *)
#endif
#if OMPI_HAVE_FORTRAN_REAL16 && OMPI_REAL16_MATCHES_C
OP_FUNC_3BUF(prod, fortran_real16, ompi_fortran_real16_t, *)
#endif
/* Complex */
#if 0
#if defined(HAVE_SHORT_FLOAT__COMPLEX)
OP_FUNC_3BUF(prod, c_short_float_complex, short float _Complex, *)
#elif defined(HAVE_OPAL_SHORT_FLOAT_COMPLEX_T)
COMPLEX_PROD_FUNC_3BUF(c_short_float_complex, opal_short_float_t)
#endif
OP_FUNC_3BUF(prod, c_float_complex, float _Complex, *)
OP_FUNC_3BUF(prod, c_double_complex, double _Complex, *)
OP_FUNC_3BUF(prod, c_long_double_complex, long double _Complex, *)
#endif // 0

/*************************************************************************
 * Logical AND
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) && (b))
/* C integer */
FUNC_FUNC_3BUF(land,   int8_t,   int8_t)
FUNC_FUNC_3BUF(land,  uint8_t,  uint8_t)
FUNC_FUNC_3BUF(land,  int16_t,  int16_t)
FUNC_FUNC_3BUF(land, uint16_t, uint16_t)
FUNC_FUNC_3BUF(land,  int32_t,  int32_t)
FUNC_FUNC_3BUF(land, uint32_t, uint32_t)
FUNC_FUNC_3BUF(land,  int64_t,  int64_t)
FUNC_FUNC_3BUF(land, uint64_t, uint64_t)
FUNC_FUNC_3BUF(land,  long,  long)
FUNC_FUNC_3BUF(land,  unsigned_long, unsigned long)

/* Logical */
#if OMPI_HAVE_FORTRAN_LOGICAL
FUNC_FUNC_3BUF(land, fortran_logical, ompi_fortran_logical_t)
#endif
/* C++ bool */
FUNC_FUNC_3BUF(land, bool, bool)

/*************************************************************************
 * Logical OR
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) || (b))
/* C integer */
FUNC_FUNC_3BUF(lor,   int8_t,   int8_t)
FUNC_FUNC_3BUF(lor,  uint8_t,  uint8_t)
FUNC_FUNC_3BUF(lor,  int16_t,  int16_t)
FUNC_FUNC_3BUF(lor, uint16_t, uint16_t)
FUNC_FUNC_3BUF(lor,  int32_t,  int32_t)
FUNC_FUNC_3BUF(lor, uint32_t, uint32_t)
FUNC_FUNC_3BUF(lor,  int64_t,  int64_t)
FUNC_FUNC_3BUF(lor, uint64_t, uint64_t)
FUNC_FUNC_3BUF(lor,  long,  long)
FUNC_FUNC_3BUF(lor,  unsigned_long, unsigned long)

/* Logical */
#if OMPI_HAVE_FORTRAN_LOGICAL
FUNC_FUNC_3BUF(lor, fortran_logical, ompi_fortran_logical_t)
#endif
/* C++ bool */
FUNC_FUNC_3BUF(lor, bool, bool)

/*************************************************************************
 * Logical XOR
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a ? 1 : 0) ^ (b ? 1: 0))
/* C integer */
FUNC_FUNC_3BUF(lxor,   int8_t,   int8_t)
FUNC_FUNC_3BUF(lxor,  uint8_t,  uint8_t)
FUNC_FUNC_3BUF(lxor,  int16_t,  int16_t)
FUNC_FUNC_3BUF(lxor, uint16_t, uint16_t)
FUNC_FUNC_3BUF(lxor,  int32_t,  int32_t)
FUNC_FUNC_3BUF(lxor, uint32_t, uint32_t)
FUNC_FUNC_3BUF(lxor,  int64_t,  int64_t)
FUNC_FUNC_3BUF(lxor, uint64_t, uint64_t)
FUNC_FUNC_3BUF(lxor,  long,  long)
FUNC_FUNC_3BUF(lxor,  unsigned_long, unsigned long)

/* Logical */
#if OMPI_HAVE_FORTRAN_LOGICAL
FUNC_FUNC_3BUF(lxor, fortran_logical, ompi_fortran_logical_t)
#endif
/* C++ bool */
FUNC_FUNC_3BUF(lxor, bool, bool)

/*************************************************************************
 * Bitwise AND
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) & (b))
/* C integer */
FUNC_FUNC_3BUF(band,   int8_t,   int8_t)
FUNC_FUNC_3BUF(band,  uint8_t,  uint8_t)
FUNC_FUNC_3BUF(band,  int16_t,  int16_t)
FUNC_FUNC_3BUF(band, uint16_t, uint16_t)
FUNC_FUNC_3BUF(band,  int32_t,  int32_t)
FUNC_FUNC_3BUF(band, uint32_t, uint32_t)
FUNC_FUNC_3BUF(band,  int64_t,  int64_t)
FUNC_FUNC_3BUF(band, uint64_t, uint64_t)
FUNC_FUNC_3BUF(band,  long,  long)
FUNC_FUNC_3BUF(band,  unsigned_long, unsigned long)

/* Fortran integer */
#if OMPI_HAVE_FORTRAN_INTEGER
FUNC_FUNC_3BUF(band, fortran_integer, ompi_fortran_integer_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER1
FUNC_FUNC_3BUF(band, fortran_integer1, ompi_fortran_integer1_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER2
FUNC_FUNC_3BUF(band, fortran_integer2, ompi_fortran_integer2_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER4
FUNC_FUNC_3BUF(band, fortran_integer4, ompi_fortran_integer4_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER8
FUNC_FUNC_3BUF(band, fortran_integer8, ompi_fortran_integer8_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER16
FUNC_FUNC_3BUF(band, fortran_integer16, ompi_fortran_integer16_t)
#endif
/* Byte */
FUNC_FUNC_3BUF(band, byte, char)

/*************************************************************************
 * Bitwise OR
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) | (b))
/* C integer */
FUNC_FUNC_3BUF(bor,   int8_t,   int8_t)
FUNC_FUNC_3BUF(bor,  uint8_t,  uint8_t)
FUNC_FUNC_3BUF(bor,  int16_t,  int16_t)
FUNC_FUNC_3BUF(bor, uint16_t, uint16_t)
FUNC_FUNC_3BUF(bor,  int32_t,  int32_t)
FUNC_FUNC_3BUF(bor, uint32_t, uint32_t)
FUNC_FUNC_3BUF(bor,  int64_t,  int64_t)
FUNC_FUNC_3BUF(bor, uint64_t, uint64_t)
FUNC_FUNC_3BUF(bor,  long,  long)
FUNC_FUNC_3BUF(bor,  unsigned_long, unsigned long)

/* Fortran integer */
#if OMPI_HAVE_FORTRAN_INTEGER
FUNC_FUNC_3BUF(bor, fortran_integer, ompi_fortran_integer_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER1
FUNC_FUNC_3BUF(bor, fortran_integer1, ompi_fortran_integer1_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER2
FUNC_FUNC_3BUF(bor, fortran_integer2, ompi_fortran_integer2_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER4
FUNC_FUNC_3BUF(bor, fortran_integer4, ompi_fortran_integer4_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER8
FUNC_FUNC_3BUF(bor, fortran_integer8, ompi_fortran_integer8_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER16
FUNC_FUNC_3BUF(bor, fortran_integer16, ompi_fortran_integer16_t)
#endif
/* Byte */
FUNC_FUNC_3BUF(bor, byte, char)

/*************************************************************************
 * Bitwise XOR
 *************************************************************************/

#undef current_func
#define current_func(a, b) ((a) ^ (b))
/* C integer */
FUNC_FUNC_3BUF(bxor,   int8_t,   int8_t)
FUNC_FUNC_3BUF(bxor,  uint8_t,  uint8_t)
FUNC_FUNC_3BUF(bxor,  int16_t,  int16_t)
FUNC_FUNC_3BUF(bxor, uint16_t, uint16_t)
FUNC_FUNC_3BUF(bxor,  int32_t,  int32_t)
FUNC_FUNC_3BUF(bxor, uint32_t, uint32_t)
FUNC_FUNC_3BUF(bxor,  int64_t,  int64_t)
FUNC_FUNC_3BUF(bxor, uint64_t, uint64_t)
FUNC_FUNC_3BUF(bxor,  long,  long)
FUNC_FUNC_3BUF(bxor,  unsigned_long, unsigned long)

/* Fortran integer */
#if OMPI_HAVE_FORTRAN_INTEGER
FUNC_FUNC_3BUF(bxor, fortran_integer, ompi_fortran_integer_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER1
FUNC_FUNC_3BUF(bxor, fortran_integer1, ompi_fortran_integer1_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER2
FUNC_FUNC_3BUF(bxor, fortran_integer2, ompi_fortran_integer2_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER4
FUNC_FUNC_3BUF(bxor, fortran_integer4, ompi_fortran_integer4_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER8
FUNC_FUNC_3BUF(bxor, fortran_integer8, ompi_fortran_integer8_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER16
FUNC_FUNC_3BUF(bxor, fortran_integer16, ompi_fortran_integer16_t)
#endif
/* Byte */
FUNC_FUNC_3BUF(bxor, byte, char)

/*************************************************************************
 * Min and max location "pair" datatypes
 *************************************************************************/

/*
#if OMPI_HAVE_FORTRAN_REAL
LOC_STRUCT_3BUF(2real, ompi_fortran_real_t, ompi_fortran_real_t)
#endif
#if OMPI_HAVE_FORTRAN_DOUBLE_PRECISION
LOC_STRUCT_3BUF(2double_precision, ompi_fortran_double_precision_t, ompi_fortran_double_precision_t)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER
LOC_STRUCT_3BUF(2integer, ompi_fortran_integer_t, ompi_fortran_integer_t)
#endif
LOC_STRUCT_3BUF(float_int, float, int)
LOC_STRUCT_3BUF(double_int, double, int)
LOC_STRUCT_3BUF(long_int, long, int)
LOC_STRUCT_3BUF(2int, int, int)
LOC_STRUCT_3BUF(short_int, short, int)
LOC_STRUCT_3BUF(long_double_int, long double, int)
*/

/*************************************************************************
 * Max location
 *************************************************************************/

#if OMPI_HAVE_FORTRAN_REAL
LOC_FUNC_3BUF(maxloc, 2real, >)
#endif
#if OMPI_HAVE_FORTRAN_DOUBLE_PRECISION
LOC_FUNC_3BUF(maxloc, 2double_precision, >)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER
LOC_FUNC_3BUF(maxloc, 2integer, >)
#endif
LOC_FUNC_3BUF(maxloc, float_int, >)
LOC_FUNC_3BUF(maxloc, double_int, >)
LOC_FUNC_3BUF(maxloc, long_int, >)
LOC_FUNC_3BUF(maxloc, 2int, >)
LOC_FUNC_3BUF(maxloc, short_int, >)
LOC_FUNC_3BUF(maxloc, long_double_int, >)

/*************************************************************************
 * Min location
 *************************************************************************/

#if OMPI_HAVE_FORTRAN_REAL
LOC_FUNC_3BUF(minloc, 2real, <)
#endif
#if OMPI_HAVE_FORTRAN_DOUBLE_PRECISION
LOC_FUNC_3BUF(minloc, 2double_precision, <)
#endif
#if OMPI_HAVE_FORTRAN_INTEGER
LOC_FUNC_3BUF(minloc, 2integer, <)
#endif
LOC_FUNC_3BUF(minloc, float_int, <)
LOC_FUNC_3BUF(minloc, double_int, <)
LOC_FUNC_3BUF(minloc, long_int, <)
LOC_FUNC_3BUF(minloc, 2int, <)
LOC_FUNC_3BUF(minloc, short_int, <)
LOC_FUNC_3BUF(minloc, long_double_int, <)
